
#include <hip/hip_runtime.h>
extern "C" {
    
#ifndef DTYPE
#define DTYPE float
#endif

    __global__ void tensor_1d_equals (const int n,
                                      const DTYPE* x, const int offset_x, const int stride_x,
                                      const DTYPE* y, const int offset_y, const int stride_y,
                                      int* eq_flag) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = offset_x + gid * stride_x;
            const int iy = offset_y + gid * stride_y;
            if (x[ix] != y[iy]) {
                eq_flag[0]++;
            }
        }
    }
    
    __global__ void tensor_2d_equals (const int n, const int c,
                                      const DTYPE* x, const int offset_x, const int n_x, const int c_x,
                                      const DTYPE* y, const int offset_y, const int n_y, const int c_y,
                                      int* eq_flag) {
        const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_n < n) && (gid_c < c);
        if (valid) {
            const int ix = offset_x + gid_n * n_x + gid_c * c_x;
            const int iy = offset_y + gid_n * n_y + gid_c * c_y;
            if (x[ix] != y[iy]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void tensor_3d_equals (const int n, const int c, const int h,
                                      const DTYPE* x, const int offset_x,
                                      const int n_x, const int c_x, const int h_x,
                                      const DTYPE* y, const int offset_y,
                                      const int n_y, const int c_y, const int h_y,
                                      int* eq_flag) {
        const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
        const int gid_h = blockIdx.z * blockDim.z + threadIdx.z;
        const bool valid = (gid_n < n) && (gid_c < c) && (gid_h < h);
        if (valid) {
            const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_h * h_x;
            const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_h * h_y;
            if (x[ix] != y[iy]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void tensor_4d_equals (const int n, const int c, const int h, const int w,
                                      const DTYPE* x, const int offset_x,
                                      const int n_x, const int c_x, const int h_x, const int w_x,
                                      const DTYPE* y, const int offset_y,
                                      const int n_y, const int c_y, const int h_y, const int w_y,
                                      int* eq_flag) {
        const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
        const int gid_h = blockIdx.z * blockDim.z + threadIdx.z;
        const bool valid = (gid_n < n) && (gid_c < c) && (gid_h < h);
        if (valid) {
            const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_h * h_x;
            const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_h * h_y;
            for (int i = 0; i < w; i++) {
                if (x[ix + i * w_x] != y[iy + i * w_y]){
                    eq_flag[0]++;
                }
            };
        }
    }

    __global__ void tensor_5d_equals (const int n, const int c, const int d, const int h, const int w,
                                      const DTYPE* x, const int offset_x,
                                      const int n_x, const int c_x, const int d_x, const int h_x, const int w_x,
                                      const DTYPE* y, const int offset_y,
                                      const int n_y, const int c_y, const int d_y, const int h_y, const int w_y,
                                      int* eq_flag) {
        const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
        const int gid_d = blockIdx.z * blockDim.z + threadIdx.z;
        const bool valid = (gid_n < n) && (gid_c < c) && (gid_d < d);
        if (valid) {
            const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_d * d_x;
            const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_d * d_y;
            for (int i = 0; i < h; i++) {
                for (int j = 0; j < w; j++) {
                    if (x[ix + i * h_x + j * w_x] != y[iy + i * h_y + j * w_y]){
                        eq_flag[0]++;
                    }
                }
            };
        }
    }

}
